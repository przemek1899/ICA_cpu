#include "hip/hip_runtime.h"

/*
 * PCA Principal Component Analysis on raw data
 * This implementation bases on matlab pca implementation
 */

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""

__global__ void pca_gpu(){

}

void runPCA(void){

	checkCudaErrors(hipSetDevice(0));
	/*
	float *dev_A, *dev_C;
	//allocate memory
    checkCudaErrors(hipMalloc(&dev_A, m*n*sizeof(float)));
    checkCudaErrors(hipMalloc(&dev_C, m*m*sizeof(float)));

	// copy data from cpu to gpu memory
    checkCudaErrors(hipMemcpy(dev_A, A, m*n*sizeof(float), hipMemcpyHostToDevice));
	*/

	hipEvent_t start, stop;
	float elapsedTime;

	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	checkCudaErrors(hipEventRecord(start, 0));

	// call kernel function here

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));


	/*
	//copy results from gpu memory to cpu
	checkCudaErrors(hipMemcpy(C, dev_C, m*m*sizeof(float), hipMemcpyDeviceToHost));

	//free gpu memory
    checkCudaErrors(hipFree(dev_C));
    checkCudaErrors(hipFree(dev_A));
    */

	checkCudaErrors(hipDeviceReset()); // dla debuggera

	printf("Kernel-only time: %f ms\n", elapsedTime);

	return;
}