#include "hip/hip_runtime.h"
/*
 * PCA Principal Component Analysis on raw data
 * This implementation bases on matlab pca implementation
 */

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include <cula_lapack_device.h>
#include "pca.cuh"
#include <fstream>
#include <iostream>

#define imin(X, Y)  ((X) < (Y) ? (X) : (Y))
#define imax(X, Y)  ((X) > (Y) ? (X) : (Y))
#define NUM_COMPONENTS = 20;

void checkStatus(culaStatus status)
{
    char buf[256];
    if(!status)
        return;
    culaGetErrorInfoString(status, culaGetErrorInfo(), buf, sizeof(buf));
    printf("%s\n", buf);
    culaShutdown();
    exit(EXIT_FAILURE);
}

int getRound(int m, int n){

	if (m % n == 0)
		return m;
	else
		return (m/n) * n + n;
}

__device__ int deviceGetRound(int m, int n){

	if (m % n == 0)
		return m;
	return (m/n) * n + n;
}

/*

colsign2 functions

*/

__global__ void colsign2(nifti_data_type* coeff, int rows, int cols, nifti_data_type * intermediate_results, int m_colsign, int n_colsign){

	extern __shared__ nifti_data_type Ash[];
	int tid = threadIdx.x;

	Ash[tid] = 0.0;
	int x_global_index = threadIdx.x + blockIdx.x * blockDim.x;// + i*gridDim.x;
	if (blockIdx.y < cols && x_global_index < rows){
		int global_data_index = x_global_index + blockIdx.y * rows;

		// find max(abs)
		// shared memory version
		Ash[tid] = coeff[global_data_index];
		__syncthreads();

		int result;
		for (unsigned int s = blockDim.x/2; s>0; s>>=1){
			if (tid < s){
				//Ash[tid] += Ash[tid+s];
				nifti_data_type a = Ash[tid]; nifti_data_type b = Ash[tid+s];
				result = (fabs(a)-fabs(b))>0;
				Ash[tid] = result * a + fabs((nifti_data_type)result - 1) * b;
			}
			__syncthreads();
		}
			
		if (tid == 0){

			intermediate_results[blockIdx.x + blockIdx.y*gridDim.x] = Ash[0];
			//(r == 0)*(-1) + (r > 0);
		}
	}
}

__global__ void get_colsign(nifti_data_type *intmed_results, int rows, int cols, nifti_data_type * coeff, int m_coeff, int n_coeff, nifti_data_type* maxFindResults_d){

	extern __shared__ nifti_data_type Ash[];

	int tid = threadIdx.x;
	
	Ash[tid] = 0.0;
	if (blockIdx.x < cols){
		Ash[tid] = intmed_results[tid + rows*blockIdx.x];
	}

	// find max(abs)
	int result;
	for (unsigned int s = blockDim.x/2; s>0; s>>=1){
		if (tid < s){
			//Ash[tid] += Ash[tid+s];
			nifti_data_type a = Ash[tid]; nifti_data_type b = Ash[tid+s];
			result = (fabs(a)-fabs(b))>0;
			Ash[tid] = result * a + fabs((nifti_data_type)result - 1) * b;
		}
		__syncthreads();
	}

	int r = Ash[0] >= 0;
	int sign = (r == 0)*(-1) + (r > 0);
	if (tid == 0 && blockIdx.x < cols){
		maxFindResults_d[blockIdx.x] = sign;
	}

	// do tego momentu jest dobrze

	//jeden blok - jedna kolumna (w macierzy coeff, wynikowej U z svd, kt�ra siedzi w tablicy A)
	if (blockIdx.x < cols){
		int iter = deviceGetRound(m_coeff, blockDim.x) / blockDim.x; // m / d�ugo�� bloku
		for (unsigned i=0; i < iter; i++){
			//int index = tid + blockDim.x*blockIdx.x + i * gridDim.x;
			int rowIndex = tid + blockDim.x * i;
			if (rowIndex < m_coeff){
				coeff[rowIndex + blockIdx.x*m_coeff] *= sign;
			}
		}
	}
}


void print_matrix_data(float * Matrix, int m, int n, int print_to_shell, int write_to_file, const char * filename){

	if (write_to_file && print_to_shell){
		std::ofstream file_data;
		file_data.open(filename);

		for(int i=0; i < n; i++){
			std::cout << Matrix[i] << std::endl;
			file_data << Matrix[i] << "\n";
		}

		file_data.close();
	}
	else if(write_to_file){
		std::ofstream file_data;
		file_data.open(filename);

		if (n == 0){
			for(int j=0; j<m; j++){
				file_data << Matrix[j] << "\n";
			}
		}
		else{
			for(int i=0; i < n; i++){
				for(int j=0; j<m; j++){
					file_data << Matrix[i*m +j] << " ";
				}
				file_data << "\n";
			}
		}
		file_data.close();
	}
	else if(print_to_shell){
		for(int i=0; i < n; i++){
			std::cout << Matrix[i] << std::endl;
		}
	}

	return;
}

__device__ inline double __shfl_down_double(double var, unsigned int srcLane, int width=32) {
  int2 a = *reinterpret_cast<int2*>(&var);
  a.x = __shfl_down(a.x, srcLane, width);
  a.y = __shfl_down(a.y, srcLane, width);
  return *reinterpret_cast<double*>(&a);
}

__device__ inline double __sfhl_down_asm_double(double var, unsigned int delta, int width=32){

	int lo, hi;
	asm volatile("mov.b64 {%0, %1}, %2;" : "=r"(lo), "=r" (hi) : "d"(var));

	// shuffle the 32b registers
	lo = __shfl_down(lo, delta, width);
	hi = __shfl_down(hi, delta, width);

	// recreate the 64b number
	asm volatile("mov.b64 %0, {%1, %2};" : "=d"(var) : "r"(lo), "r"(hi));

	return var;
}

__device__ __inline__ double shfl_asm_double(double x, int laneId){

	// Split the double number into 2 32b registers
	int lo, hi;
	asm volatile("mov.b64 {%0, %1}, %2;" : "=r"(lo), "=r" (hi) : "d"(x));

	// shuffle the 32b registers
	lo = __shfl(lo, laneId);
	hi = __shfl(hi, laneId);

	// recreate the 64b number
	asm volatile("mov.b64 %0, {%1, %2};" : "=d"(x) : "r"(lo), "r"(hi));

	return x;
}

__inline__ __device__ double warpDoubleReduce(double val){
	
	for (int offset = warpSize/2; offset > 0; offset /= 2){
		val += __shfl_down_double(val, offset);
	}
	return val;
}

__inline__ __device__ nifti_data_type blockReduceMuColumn(nifti_data_type val){
	
  static __shared__ nifti_data_type shared[32]; // Shared mem for 32 partial sums
  int lane = threadIdx.x % warpSize;
  int wid = threadIdx.x / warpSize;

  val = warpDoubleReduce(val);

  if (lane==0) shared[wid]=val;
  __syncthreads();

  //read from shared memory only if that warp existed
  val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;
  if (wid==0) val = warpDoubleReduce(val); //Final reduce within first warp

  return val;
}

__global__ void mu_shuffle(nifti_data_type * A, int m, int n, int iter, nifti_data_type* MU){

	// na razie zakladamy, ze jeden blok mo�e przykry� ca�� kolumn�, czyli liczba element�w w kolumnie <= 1024

	static __shared__ nifti_data_type shared[32]; // Shared mem for 32 partial sums
	int lane = threadIdx.x % warpSize;
	int wid = threadIdx.x / warpSize;

	int tid = threadIdx.x;
	int difference = blockDim.x - m;
	int grid_index = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i=0; i<iter; i++){
		int columnIndex = blockIdx.x + i * gridDim.x;
		int globalDataIndex = grid_index - blockIdx.x * difference + i * gridDim.x * m;
		
		if (columnIndex < n){
			// do reduction by shuffle instructions in warps
			nifti_data_type val = 0.0;
			if (tid < m){
				val = A[globalDataIndex];
			}
			__syncthreads(); 
			
			val = warpDoubleReduce(val);
			if (lane==0) shared[wid]=val;
			__syncthreads(); 
			
			//read from shared memory only if that warp existed
			val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;

			if (wid==0) val = warpDoubleReduce(val); //Final reduce within first warp

			nifti_data_type mean = val / m;
			if (tid < m){
				A[globalDataIndex] -= mean;
			}
			if (tid == 0 ){
				MU[columnIndex] = mean;
			}
		}
	}
}

/*	MU MU MU MU MU MU MU MU MU MU MU MU MU MU MU MU MU MU MU MU

	w tej wersji funkcji get_mu oblicza tablice jak ponizej:

						M = 163840 (to jest problem - du�a liczba kolumn)
		_________________________________________________________
		|___|___|___|___|___|___|___|___|___|___|___|___|___|___|
		|___|___|___|___|___|___|___|___|___|___|___|___|___|___|
  N=121 |___|___|___|___|___|___|___|___|___|___|___|___|___|___|
		|___|___|___|___|___|___|___|___|___|___|___|___|___|___|
		|___|___|___|___|___|___|___|___|___|___|___|___|___|___|

		Ka�dy BLOK wykonuje redukcj� sumy dla JEDNEJ KOLUMNY

		PROBLEM: liczba kolumn jest zbyt du�y aby "przykry�" ca�� tablic� blokami za jednym razem - dlatego potrzebne s� iteracje
		ALE: co je�li liczba wierszy (b�dzie wi�ksza od 128 albo np. 1024 ??) - to do przemy�lenia

*/

__global__ void get_mu(nifti_data_type * A, int m, int n, int iter, nifti_data_type* MU){

	// tutaj jest zalozenie z m < n (np. m=121, n=163840)
	// every block has an array (shared memory) of ceil((threads_per_blocks / warpSize)) elements
	
	extern __shared__ nifti_data_type Ash[];
	int tid = threadIdx.x;
	int difference = blockDim.x - m;
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	//#pragma unroll
	for (int i=0; i<iter; i++){
		
		int columnIndex = blockIdx.x + i * gridDim.x;
		int globalDataIndex = index - blockIdx.x * difference + i * gridDim.x * m;
		if (columnIndex < n){
			
			Ash[tid] = 0.0; // initialize all to zeros (padding the rest of elements which are not part of array
			// each thread loads one element from global memory to shared memory
			if (tid < m){
				Ash[tid] = A[globalDataIndex];
			}
			__syncthreads();

			// do reduction in shared memory
			for (unsigned int s = blockDim.x/2; s>0; s>>=1){
				if (tid < s){
					Ash[tid] += Ash[tid+s];
				}
				__syncthreads();
			}
			
			nifti_data_type mean = Ash[0] / m;
			if (tid < m){
				A[globalDataIndex] -= mean;
			}
			if (tid == 0 ){
				MU[columnIndex] = mean;
			}
		}
	}
	
}

void runPCA(nifti_data_type * A, int m, int n){

	// int DOF = n - 1;
		
	/* Initialize CULA */
	checkCudaErrors(hipSetDevice(0));
    culaStatus status;
    status = culaInitialize();
    checkStatus(status);
	
	// for m >= n
	char jobu = 'O';  // n > m ? 'S' : 'O';
	char jobvt = 'N'; // n > m ? 'O' : 'S'; bylo S

	// for n > m 
	if (n > m){
		jobu = 'N';
		jobvt = 'O';
	}
	
	const int NCOMPONENTS = 20;
	int lda = m;
    int ldu = m;
    int ldvt = n;
	int min = imin(m,n);
	int max = imax(m,n);
	hipEvent_t start, stop;
	float elapsedTime;

	nifti_data_type *S, *U, *VT;
	nifti_data_type *A_dev, *AT_dev, *MU_dev, *S_dev, *U_dev, *VT_dev;

	// allocation of memory
	checkCudaErrors(hipMalloc(&A_dev, m*n*sizeof(nifti_data_type)));
	checkCudaErrors(hipMalloc(&AT_dev, m*n*sizeof(nifti_data_type))); // AT do transpozycji macierzy
	checkCudaErrors(hipMalloc(&MU_dev, m*sizeof(nifti_data_type))); // an array only for check the results, need to be removed in final version
		
	S = (nifti_data_type*) malloc(min * sizeof(nifti_data_type));
	checkCudaErrors(hipMalloc(&S_dev, min * sizeof(nifti_data_type)));

	if (jobu != 'O' && jobu != 'N'){
		U = (nifti_data_type*) calloc(ldu*m, sizeof(nifti_data_type));
		checkCudaErrors(hipMalloc(&U_dev, ldu*m*sizeof(nifti_data_type)));
	}
	if (jobvt != 'O' && jobvt != 'N'){
		VT = (nifti_data_type*) malloc(ldvt*n*sizeof(nifti_data_type));
		checkCudaErrors(hipMalloc(&VT_dev, ldvt*n*sizeof(nifti_data_type)));
	}


	// copy data from host to device
	checkCudaErrors(hipMemcpy(A_dev, A, m*n*sizeof(nifti_data_type), hipMemcpyHostToDevice));

	// transpozycja macierzy A w celu obliczenia mu
	status = culaDeviceSgeTranspose(m, n, A_dev, m, AT_dev, n);
    checkStatus(status);

	//printf("Calculete transpose-only time: %f ms\n", elapsedTime);
	
	// ---------- MU calculations -----------------------
	int shared_mem_size = getRound(min, 32)*sizeof(nifti_data_type);
	int threadsPerBlock = 128;
	int numBlocks = 65535;
	int iter = getRound(m, numBlocks) / numBlocks;
	printf("shared mem size %d, iter %d\n", shared_mem_size, iter);

	
	//get_mu<<<numBlocks, threadsPerBlock, shared_mem_size>>>(AT_dev, n, m, iter, MU_dev);
	mu_shuffle<<<numBlocks, threadsPerBlock>>>(AT_dev, n, m, iter, MU_dev);
	checkCudaErrors(hipDeviceSynchronize()); checkCudaErrors(hipGetLastError());

	// transpozycja macierzy AT po obliczenia mu
	status = culaDeviceSgeTranspose(n,  m, AT_dev, n, A_dev, m);
    checkStatus(status);

	nifti_data_type *MU = (nifti_data_type*) malloc(max*sizeof(nifti_data_type));
	checkCudaErrors(hipMemcpy(MU, MU_dev, max*sizeof(nifti_data_type), hipMemcpyDeviceToHost));
	print_matrix_data(MU, 1, m, 0, 1, "mu_shuffle.txt");
	free(MU);

	// ------------- SVD -----------------------------
	// coeff = U_dev (m x min)
	
	checkCudaErrors(hipEventCreate(&start));	checkCudaErrors(hipEventCreate(&stop));	checkCudaErrors(hipEventRecord(start, 0));

    status = culaDeviceSgesvd(jobu, jobvt, m, n, A_dev, lda, S_dev, U_dev, ldu, VT_dev, ldvt);
    checkStatus(status);
		
	checkCudaErrors(hipEventRecord(stop, 0));	checkCudaErrors(hipEventSynchronize(stop));	
	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("Calculate svd time: %f ms\n", elapsedTime);
	//checkCudaErrors(hipMemcpy(S, S_dev, min*sizeof(nifti_data_type), hipMemcpyDeviceToHost));
	//print_matrix_data(S, min, 0, 0, 1, "S_matrix.txt");

	// -------------- colsign ---------------------------------
	threadsPerBlock = 512;
	int blocks_per_column = getRound(m, threadsPerBlock) / threadsPerBlock;
	int grid_x = getRound(NCOMPONENTS, 32);
	dim3 grid(blocks_per_column, grid_x);
	shared_mem_size = threadsPerBlock*sizeof(nifti_data_type);
	printf("shared mem size %d, iter %d\n", shared_mem_size);

	nifti_data_type * intermediate_results;
	checkCudaErrors(hipMalloc(&intermediate_results, blocks_per_column*NCOMPONENTS*sizeof(nifti_data_type)));

	colsign2<<<grid, threadsPerBlock, shared_mem_size>>>(A_dev, m, NCOMPONENTS, intermediate_results, blocks_per_column, NCOMPONENTS);
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipGetLastError());

	//nifti_data_type * intermediate_results_h = (nifti_data_type*) malloc(new_cols*blocks_per_column*sizeof(nifti_data_type));
	//checkCudaErrors(hipMemcpy(intermediate_results_h, intermediate_results, new_cols*blocks_per_column*sizeof(nifti_data_type), hipMemcpyDeviceToHost));
	//print_matrix_data(intermediate_results_h, blocks_per_column, new_cols, 0, 1, "intermed_res.txt");
	//free(intermediate_results_h);

	// ------------------- get_colsign ----------------------------------------------------
	nifti_data_type* maxFindResults = (nifti_data_type*) malloc(NCOMPONENTS*sizeof(nifti_data_type));
	nifti_data_type* maxFindResults_d;
	checkCudaErrors(hipMalloc(&maxFindResults_d, NCOMPONENTS*sizeof(nifti_data_type)));

	dim3 grid2(grid_x, 1);
	shared_mem_size = blocks_per_column*sizeof(nifti_data_type);
	get_colsign<<<grid2, blocks_per_column, shared_mem_size>>>(intermediate_results, blocks_per_column, NCOMPONENTS, A_dev, m, NCOMPONENTS, maxFindResults_d);
	
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipGetLastError());

	nifti_data_type* coeff = (nifti_data_type*) malloc(m*NCOMPONENTS*sizeof(nifti_data_type));
	checkCudaErrors(hipMemcpy(coeff, A_dev, m*NCOMPONENTS*sizeof(nifti_data_type), hipMemcpyDeviceToHost));
	print_matrix_data(coeff, m, NCOMPONENTS, 0, 1, "coeff_mat.txt");

	checkCudaErrors(hipMemcpy(maxFindResults, maxFindResults_d, NCOMPONENTS*sizeof(nifti_data_type), hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(maxFindResults_d));
	print_matrix_data(maxFindResults, NCOMPONENTS, 0, 0, 1, "max_results.txt");
	free(maxFindResults);

	// free memory
	free(S);
	free(coeff);
	checkCudaErrors(hipFree(intermediate_results));
	checkCudaErrors(hipFree(A_dev));
	checkCudaErrors(hipFree(S_dev));
	checkCudaErrors(hipFree(MU_dev));
	checkCudaErrors(hipFree(AT_dev));

	if (jobu != 'O' && jobu != 'N'){
		free(U);
		checkCudaErrors(hipFree(U_dev));
	}
	if (jobvt != 'O' && jobvt != 'N'){
		free(VT);
		checkCudaErrors(hipFree(VT_dev));
	}

	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));

	//checkCudaErrors(hipDeviceReset()); // dla debuggera
	return;
}