#include "hip/hip_runtime.h"
/*
 * PCA Principal Component Analysis on raw data
 * This implementation bases on matlab pca implementation
 */

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include <cula_lapack_device.h>
#include "pca.cuh"
#include <fstream>

#define imin(X, Y)  ((X) < (Y) ? (X) : (Y))

int getRound(int m, int n){

	if (m % n == 0)
		return m;
	else
		return (m/n) * n + n;
}


__device__ __inline__ double shfl_double(double x, int laneId){

	// Split the double number into 2 32b registers
	int lo, hi;
	asm volatile("mov.b32 {%0, %1}, %2;" : "=r"(lo), "=r" (hi) : "d"(x));

	// shuffle the 32b registers
	lo = __shfl(lo, laneId);
	hi = __shfl(hi, laneId);

	// recreate the 64b number
	asm volatile("mov.b64 %0, {%1, %2};" : "=d"(x) : "r"(lo), "r"(hi));

	return x;
}

__global__ void mu_shuffle(nifti_data_type * A, int m, int n, int iter){

}

__global__ void test_shuffle_reduce() {

	int laneId = threadIdx.x & 0x1f;
	int value = 31 - laneId;

	// Use XOR to perform butterfly shuffle
	for(unsigned int i=16; i>=1; i/=2){
		value += __shfl_xor(value, i, 32);
	}
	// "value" now contains the sum across all threads 
	printf("Thread %d final value = %d\n", threadIdx.x, value);
}


__global__ void get_mu(nifti_data_type * A, int m, int n, int iter){

	// in this version thera are not yet weights, not needed now
	extern __shared__ nifti_data_type Ash[];
	int tid = threadIdx.x;
	int difference = blockDim.x - m;
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	//#pragma unroll
	for (int i=0; i<iter; i++){
		
		int columnIndex = blockIdx.x + i * gridDim.x;
		int globalDataIndex = index - blockIdx.x * difference + i * gridDim.x * m;
		if (columnIndex < n){

			Ash[tid] = 0.0; // initialize all to zeros (padding the rest of elements which are not part of array
			// each thread loads one element from global memory to shared memory
			if (tid < m){
				Ash[tid] = A[globalDataIndex];
			}
			__syncthreads();

			// do reduction in shared memory
			for (unsigned int s = blockDim.x/2; s>0; s>>=1){
				if (tid < s){
					Ash[tid] += Ash[tid+s];
				}
				__syncthreads();
			}

			int mean = Ash[0] / m;
			if (tid < m){
				A[globalDataIndex] -= mean;
			}
		}
	}
	
}

void checkStatus(culaStatus status)
{
    char buf[256];
    if(!status)
        return;
    culaGetErrorInfoString(status, culaGetErrorInfo(), buf, sizeof(buf));
    printf("%s\n", buf);
    culaShutdown();
    exit(EXIT_FAILURE);
}

void runPCA(nifti_data_type * A, int m, int n){

	// int DOF = n - 1;
		
	/* Initialize CULA */
	checkCudaErrors(hipSetDevice(0));
    culaStatus status;
    status = culaInitialize();
    checkStatus(status);
	
	// for m >= n
	char jobu = 'O';  // n > m ? 'S' : 'O';
	char jobvt = 'S'; // n > m ? 'O' : 'S';

	// for n > m 
	if (n > m){
		jobu = 'S';
		jobvt = 'O';
	}

	int lda = m;
    int ldu = m;
    int ldvt = n;
	int min = imin(m,n);	
	int i;

	test_shuffle_reduce<<< 1, 32 >>>(); 
	hipDeviceSynchronize(); 
	checkCudaErrors(hipGetLastError());

	nifti_data_type *S, *U, *VT;
	nifti_data_type *A_dev, *MU_dev, *S_dev, *U_dev, *VT_dev;

	checkCudaErrors(hipMalloc(&A_dev, m*n*sizeof(nifti_data_type)));
	checkCudaErrors(hipMemcpy(A_dev, A, m*n*sizeof(nifti_data_type), hipMemcpyHostToDevice));

	/* obliczanie warto�ci mu */
	//checkCudaErrors(hipMalloc(&MU_dev, n*sizeof(nifti_data_type)));

	int shared_mem_size = getRound(m, 32)*sizeof(nifti_data_type);
	int threadsPerBlock = 128;
	int numBlocks = 65535;
	int iter = getRound(n, numBlocks) / numBlocks;
	printf("shared memory size is %d and iter %d\n", shared_mem_size, iter);

	hipEvent_t start, stop;
	float elapsedTime;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	checkCudaErrors(hipEventRecord(start, 0));

	get_mu<<<numBlocks, threadsPerBlock, shared_mem_size>>>(A_dev, m, n, iter);

	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));

	checkCudaErrors(hipGetLastError());
	/*
	//sprawdzenie warto�ci - kopiowanie do cpu - to w przyszlosci zostanie usuni�te
	//nifti_data_type *MU = (nifti_data_type*) malloc(n*sizeof(nifti_data_type));
	//checkCudaErrors(hipMemcpy(MU, MU_dev, n*sizeof(nifti_data_type), hipMemcpyDeviceToHost));
	
	// reading & writing mu array - print_matrix_data(MU, n, 0, 1, "mu_file.txt");

	free(MU);
	/* koniec obliczania warto�ci mu */

	S = (nifti_data_type*) malloc(min * sizeof(nifti_data_type));
	checkCudaErrors(hipMalloc(&S_dev, min * sizeof(nifti_data_type)));

	if (jobu != 'O' && jobu != 'N'){
		U = (nifti_data_type*) calloc(ldu*m, sizeof(nifti_data_type));
		checkCudaErrors(hipMalloc(&U_dev, ldu*m*sizeof(nifti_data_type)));
	}
	if (jobvt != 'O' && jobvt != 'N'){
		VT = (nifti_data_type*) malloc(ldvt*n*sizeof(nifti_data_type));
		checkCudaErrors(hipMalloc(&VT_dev, ldvt*n*sizeof(nifti_data_type)));
	}

	/* Perform singular value decomposition CULA */
    printf("Performing singular value decomposition using CULA ... ");

    status = culaDeviceDgesvd(jobu, jobvt, m, n, A_dev, lda, S_dev, U_dev, ldu, VT_dev, ldvt);
    checkStatus(status);

	checkCudaErrors(hipGetLastError());

	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));

	checkCudaErrors(hipMemcpy(S, S_dev, min*sizeof(nifti_data_type), hipMemcpyDeviceToHost));
	
	// reading S Matrinx - print_matrix_data(S, min, 1, 1, "Smatrix.txt")

	//free memory
	free(S);
	checkCudaErrors(hipFree(A_dev));
	checkCudaErrors(hipFree(S_dev));
	//checkCudaErrors(hipFree(MU_dev));

	if (jobu != 'O' && jobu != 'N'){
		free(U);
		checkCudaErrors(hipFree(U_dev));
	}
	if (jobvt != 'O' && jobvt != 'N'){
		free(VT);
		checkCudaErrors(hipFree(VT_dev));
	}

	checkCudaErrors(hipDeviceReset()); // dla debuggera
	
	printf("Calculete mu-only time: %f ms\n", elapsedTime);
	return;
}
