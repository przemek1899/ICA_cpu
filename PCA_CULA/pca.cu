#include "hip/hip_runtime.h"
/*
 * PCA Principal Component Analysis on raw data
 * This implementation bases on matlab pca implementation
 */

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include <cula_lapack_device.h>
#include "pca.cuh"
#include <fstream>
#include <iostream>

#define imin(X, Y)  ((X) < (Y) ? (X) : (Y))
#define imax(X, Y)  ((X) > (Y) ? (X) : (Y))

void checkStatus(culaStatus status)
{
    char buf[256];
    if(!status)
        return;
    culaGetErrorInfoString(status, culaGetErrorInfo(), buf, sizeof(buf));
    printf("%s\n", buf);
    culaShutdown();
    exit(EXIT_FAILURE);
}

int getRound(int m, int n){

	if (m % n == 0)
		return m;
	else
		return (m/n) * n + n;
}

__device__ int deviceGetRound(int m, int n){

	if (m % n == 0)
		return m;
	return (m/n) * n + n;
}

/*

sign_convention1 function searches for an element in a column (row) with the biggest absolute value,
than wirtes each found element to an array which later is used by another kernel program

*/

__global__ void sign_convention1(nifti_data_type* coeff, int rows, int cols, nifti_data_type * intermediate_results, int m_colsign, int n_colsign){

	extern __shared__ nifti_data_type Ash[];
	int tid = threadIdx.x;

	Ash[tid] = 0.0;
	int x_global_index = threadIdx.x + blockIdx.x * blockDim.x;// + i*gridDim.x;
	if (blockIdx.y < cols && x_global_index < rows){
		int global_data_index = x_global_index + blockIdx.y * rows;

		// find max(abs)
		// shared memory version
		Ash[tid] = coeff[global_data_index];
		__syncthreads();

		int result;
		for (unsigned int s = blockDim.x/2; s>0; s>>=1){
			if (tid < s){
				//Ash[tid] += Ash[tid+s];
				nifti_data_type a = Ash[tid]; nifti_data_type b = Ash[tid+s];
				result = (fabs(a)-fabs(b))>0;
				Ash[tid] = result * a + fabs((nifti_data_type)result - 1) * b;
			}
			__syncthreads();
		}
			
		if (tid == 0){

			intermediate_results[blockIdx.x + blockIdx.y*gridDim.x] = Ash[0];
			//(r == 0)*(-1) + (r > 0);
		}
	}
}

__global__ void sign_convention2(nifti_data_type *intmed_results, int rows, int cols, nifti_data_type * coeff, int m_coeff, int n_coeff, nifti_data_type* maxFindResults_d){

	extern __shared__ nifti_data_type Ash[];

	int tid = threadIdx.x;
	
	Ash[tid] = 0.0;
	if (blockIdx.x < cols){
		Ash[tid] = intmed_results[tid + rows*blockIdx.x];
	}

	// find max(abs)
	int result;
	for (unsigned int s = blockDim.x/2; s>0; s>>=1){
		if (tid < s){
			//Ash[tid] += Ash[tid+s];
			nifti_data_type a = Ash[tid]; nifti_data_type b = Ash[tid+s];
			result = (fabs(a)-fabs(b))>0;
			Ash[tid] = result * a + fabs((nifti_data_type)result - 1) * b;
		}
		__syncthreads();
	}

	int r = Ash[0] >= 0;
	int sign = (r == 0)*(-1) + (r > 0);
	/*
	if (tid == 0 && blockIdx.x < cols){
		maxFindResults_d[blockIdx.x] = sign;
	}*/

	if (blockIdx.x < cols){
		int iter = deviceGetRound(m_coeff, blockDim.x) / blockDim.x; // m / d�ugo�� bloku
		for (unsigned i=0; i < iter; i++){
			//int index = tid + blockDim.x*blockIdx.x + i * gridDim.x;
			int rowIndex = tid + blockDim.x * i;
			if (rowIndex < m_coeff){
				coeff[rowIndex + blockIdx.x*m_coeff] *= sign;
			}
		}
	}
}


void print_matrix_data(float * Matrix, int m, int n, int print_to_shell, int write_to_file, const char * filename){

	if (write_to_file && print_to_shell){
		std::ofstream file_data;
		file_data.open(filename);

		for(int i=0; i < n; i++){
			std::cout << Matrix[i] << std::endl;
			file_data << Matrix[i] << "\n";
		}

		file_data.close();
	}
	else if(write_to_file){
		std::ofstream file_data;
		file_data.open(filename);

		if (n == 0){
			for(int j=0; j<m; j++){
				file_data << Matrix[j] << "\n";
			}
		}
		else{
			for(int i=0; i < n; i++){
				for(int j=0; j<m; j++){
					file_data << Matrix[i*m +j] << " ";
				}
				file_data << "\n";
			}
		}
		file_data.close();
	}
	else if(print_to_shell){
		for(int i=0; i < n; i++){
			std::cout << Matrix[i] << std::endl;
		}
	}

	return;
}

__device__ inline double __shfl_down_double(double var, unsigned int srcLane, int width=32) {
  int2 a = *reinterpret_cast<int2*>(&var);
  a.x = __shfl_down(a.x, srcLane, width);
  a.y = __shfl_down(a.y, srcLane, width);
  return *reinterpret_cast<double*>(&a);
}

__device__ inline double __sfhl_down_asm_double(double var, unsigned int delta, int width=32){

	int lo, hi;
	asm volatile("mov.b64 {%0, %1}, %2;" : "=r"(lo), "=r" (hi) : "d"(var));

	// shuffle the 32b registers
	lo = __shfl_down(lo, delta, width);
	hi = __shfl_down(hi, delta, width);

	// recreate the 64b number
	asm volatile("mov.b64 %0, {%1, %2};" : "=d"(var) : "r"(lo), "r"(hi));

	return var;
}

__device__ __inline__ double shfl_asm_double(double x, int laneId){

	// Split the double number into 2 32b registers
	int lo, hi;
	asm volatile("mov.b64 {%0, %1}, %2;" : "=r"(lo), "=r" (hi) : "d"(x));

	// shuffle the 32b registers
	lo = __shfl(lo, laneId);
	hi = __shfl(hi, laneId);

	// recreate the 64b number
	asm volatile("mov.b64 %0, {%1, %2};" : "=d"(x) : "r"(lo), "r"(hi));

	return x;
}

__inline__ __device__ double warpDoubleReduce(double val){
	
	for (int offset = warpSize/2; offset > 0; offset /= 2){
		val += __shfl_down_double(val, offset);
	}
	return val;
}

__inline__ __device__ nifti_data_type blockReduceMuColumn(nifti_data_type val){
	
  static __shared__ nifti_data_type shared[32]; // Shared mem for 32 partial sums
  int lane = threadIdx.x % warpSize;
  int wid = threadIdx.x / warpSize;

  val = warpDoubleReduce(val);

  if (lane==0) shared[wid]=val;
  __syncthreads();

  //read from shared memory only if that warp existed
  val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;
  if (wid==0) val = warpDoubleReduce(val); //Final reduce within first warp

  return val;
}

__global__ void center_data_shuffle(nifti_data_type * A, int m, int n, int iter, nifti_data_type* MU){

	// an assumption is that one row is no longer than 1024 (nframes <= 1024)

	static __shared__ nifti_data_type shared[32]; // Shared mem for 32 partial sums
	int lane = threadIdx.x % warpSize;
	int wid = threadIdx.x / warpSize;

	int tid = threadIdx.x;
	int difference = blockDim.x - m;
	int grid_index = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i=0; i<iter; i++){
		int columnIndex = blockIdx.x + i * gridDim.x;
		int globalDataIndex = grid_index - blockIdx.x * difference + i * gridDim.x * m;
		
		if (columnIndex < n){
			// do reduction by shuffle instructions in warps
			nifti_data_type val = 0.0;
			if (tid < m){
				val = A[globalDataIndex];
			}
			__syncthreads(); 
			
			val = warpDoubleReduce(val);
			if (lane==0) shared[wid]=val;
			__syncthreads(); 
			
			//read from shared memory only if that warp existed
			val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;

			if (wid==0) val = warpDoubleReduce(val); //Final reduce within first warp

			nifti_data_type mean = val / m;
			if (tid < m){
				A[globalDataIndex] -= mean;
			}
			if (tid == 0 ){
				//MU[columnIndex] = mean;
			}
		}
	}
}


__global__ void center_data(nifti_data_type * A, int m, int n, int iter, nifti_data_type* MU){

	// m < n (np. m=121, n=163840)
	// every block has an array (shared memory) of ceil((threads_per_blocks / warpSize)) elements
	
	extern __shared__ nifti_data_type Ash[];
	int tid = threadIdx.x;
	int difference = blockDim.x - m;
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	//#pragma unroll
	for (int i=0; i<iter; i++){
		
		int columnIndex = blockIdx.x + i * gridDim.x;
		int globalDataIndex = index - blockIdx.x * difference + i * gridDim.x * m;
		if (columnIndex < n){
			
			Ash[tid] = 0.0; // initialize all to zeros (padding the rest of elements which are not part of array
			// each thread loads one element from global memory to shared memory
			if (tid < m){
				Ash[tid] = A[globalDataIndex];
			}
			__syncthreads();

			// do reduction in shared memory
			for (unsigned int s = blockDim.x/2; s>0; s>>=1){
				if (tid < s){
					Ash[tid] += Ash[tid+s];
				}
				__syncthreads();
			}
			
			nifti_data_type mean = Ash[0] / m;
			if (tid < m){
				A[globalDataIndex] -= mean;
			}
			/*
			if (tid == 0 ){
				MU[columnIndex] = mean;
			}*/
		}
	}
	
}

void runPCA(nifti_data_type * A, int m, int n, int ncomponents, nifti_data_type* coeff_result){

		
	/* Initialize CULA */
	checkCudaErrors(hipSetDevice(0));
    culaStatus status;
    status = culaInitialize();
    checkStatus(status);

	hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

	int maxThreads_x = deviceProp.maxThreadsDim[0];
	
	// for m >= n
	char jobu = 'O';  // n > m ? 'S' : 'O';
	char jobvt = 'N'; // n > m ? 'O' : 'S';

	// for n > m 
	if (n > m){
		jobu = 'N';
		jobvt = 'O';
	}
	
	int lda = m;
    int ldu = m;
    int ldvt = n;
	int min = imin(m,n);
	int max = imax(m,n);
	hipEvent_t start, stop;
	float elapsedTime;

	nifti_data_type *S, *U, *VT;
	nifti_data_type *A_dev, *AT_dev, *MU_dev, *S_dev, *U_dev, *VT_dev, *intermediate_results;

	// allocation of memory
	checkCudaErrors(hipMalloc(&A_dev, m*n*sizeof(nifti_data_type)));
	checkCudaErrors(hipMalloc(&AT_dev, m*n*sizeof(nifti_data_type))); // array AT for transpose matrix
	//checkCudaErrors(hipMalloc(&MU_dev, m*sizeof(nifti_data_type))); // an array only for checking the results, need to be removed in final version
		
	//S = (nifti_data_type*) malloc(min * sizeof(nifti_data_type));
	checkCudaErrors(hipMalloc(&S_dev, min * sizeof(nifti_data_type)));

	if (jobu != 'O' && jobu != 'N'){
		U = (nifti_data_type*) calloc(ldu*m, sizeof(nifti_data_type));
		checkCudaErrors(hipMalloc(&U_dev, ldu*m*sizeof(nifti_data_type)));
	}
	if (jobvt != 'O' && jobvt != 'N'){
		VT = (nifti_data_type*) malloc(ldvt*n*sizeof(nifti_data_type));
		checkCudaErrors(hipMalloc(&VT_dev, ldvt*n*sizeof(nifti_data_type)));
	}

	checkCudaErrors(hipMemcpy(A_dev, A, m*n*sizeof(nifti_data_type), hipMemcpyHostToDevice));

	// ---------- centring the data -----------------------
	//int threadsPerBlock = 128;
	int threadsPerBlock = getRound(min, 32);
	if (threadsPerBlock > maxThreads_x){
		threadsPerBlock = maxThreads_x;
	}
	//int shared_mem_size = getRound(min, 32)*sizeof(nifti_data_type);
	int shared_mem_size = threadsPerBlock*sizeof(nifti_data_type);
	int numBlocks = 65535;
	int iter = getRound(m, numBlocks) / numBlocks;
	//printf("shared mem size %d, iter %d\n", shared_mem_size, iter);

	// transpose matrix for function to centring the data
	status = culaDeviceSgeTranspose(m, n, A_dev, m, AT_dev, n);
    checkStatus(status);
	
	center_data<<<numBlocks, threadsPerBlock, shared_mem_size>>>(AT_dev, n, m, iter, MU_dev);
	checkCudaErrors(hipDeviceSynchronize()); checkCudaErrors(hipGetLastError());

	// trasponse matrix again to restore to its initial shape
	status = culaDeviceSgeTranspose(n,  m, AT_dev, n, A_dev, m);
    checkStatus(status);

	// ------------- SVD -----------------------------
	// coeff = U_dev (m x min)
	
	//checkCudaErrors(hipEventCreate(&start));	checkCudaErrors(hipEventCreate(&stop));	checkCudaErrors(hipEventRecord(start, 0));

    status = culaDeviceSgesvd(jobu, jobvt, m, n, A_dev, lda, S_dev, U_dev, ldu, VT_dev, ldvt);
    checkStatus(status);
		
	//checkCudaErrors(hipEventRecord(stop, 0));	checkCudaErrors(hipEventSynchronize(stop));
	//checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
	//printf("Calculate svd time: %f ms\n", elapsedTime);

	// -------------- sign convention on the coefficients ---------------------------------
	threadsPerBlock = 512;
	int blocks_per_column = getRound(m, threadsPerBlock) / threadsPerBlock;
	int grid_x = getRound(ncomponents, 32);
	dim3 grid(blocks_per_column, grid_x);
	shared_mem_size = threadsPerBlock*sizeof(nifti_data_type);
	//printf("shared mem size %d, iter %d\n", shared_mem_size);

	checkCudaErrors(hipMalloc(&intermediate_results, blocks_per_column*ncomponents*sizeof(nifti_data_type)));

	sign_convention1<<<grid, threadsPerBlock, shared_mem_size>>>(A_dev, m, ncomponents, intermediate_results, blocks_per_column, ncomponents);
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipGetLastError());

	// ------------------- run second kernel ----------------------------------------------------
	//nifti_data_type* maxFindResults = (nifti_data_type*) malloc(ncomponents*sizeof(nifti_data_type));
	nifti_data_type* maxFindResults_d;
	//checkCudaErrors(hipMalloc(&maxFindResults_d, ncomponents*sizeof(nifti_data_type)));

	dim3 grid2(grid_x, 1);
	shared_mem_size = blocks_per_column*sizeof(nifti_data_type);
	sign_convention2<<<grid2, blocks_per_column, shared_mem_size>>>(intermediate_results, blocks_per_column, ncomponents, A_dev, m, ncomponents, maxFindResults_d);
	
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipGetLastError());

	//nifti_data_type* coeff = (nifti_data_type*) malloc(m*ncomponents*sizeof(nifti_data_type));
	checkCudaErrors(hipMemcpy(coeff_result, A_dev, m*ncomponents*sizeof(nifti_data_type), hipMemcpyDeviceToHost));
	//print_matrix_data(coeff, m, ncomponents, 0, 1, "coeff_mat.txt");

	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipGetLastError());

	//checkCudaErrors(hipMemcpy(maxFindResults, maxFindResults_d, ncomponents*sizeof(nifti_data_type), hipMemcpyDeviceToHost));
	//checkCudaErrors(hipFree(maxFindResults_d));
	//print_matrix_data(maxFindResults, ncomponents, 0, 0, 1, "max_results.txt");
	//free(maxFindResults);

	// free memory
	//free(S);
	//free(coeff);
	checkCudaErrors(hipFree(intermediate_results));
	checkCudaErrors(hipFree(A_dev));
	checkCudaErrors(hipFree(S_dev));
	//checkCudaErrors(hipFree(MU_dev));
	checkCudaErrors(hipFree(AT_dev));

	if (jobu != 'O' && jobu != 'N'){
		free(U);
		checkCudaErrors(hipFree(U_dev));
	}
	if (jobvt != 'O' && jobvt != 'N'){
		free(VT);
		checkCudaErrors(hipFree(VT_dev));
	}

	//checkCudaErrors(hipEventDestroy(start));
	//checkCudaErrors(hipEventDestroy(stop));

	//checkCudaErrors(hipDeviceReset());
	return;
}