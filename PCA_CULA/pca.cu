#include "hip/hip_runtime.h"

/*
 * PCA Principal Component Analysis on raw data
 * This implementation bases on matlab pca implementation
 */

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include <cula_lapack.h>
#include "pca.cuh"

#define imin(X, Y)  ((X) < (Y) ? (X) : (Y))

__global__ void pca_gpu(float* tab, int n){

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n){
		tab[i] = i*i;
	}

}

void checkStatus(culaStatus status)
{
    char buf[256];

    if(!status)
        return;

    culaGetErrorInfoString(status, culaGetErrorInfo(), buf, sizeof(buf));
    printf("%s\n", buf);

    culaShutdown();
    exit(EXIT_FAILURE);
}

void runPCA(nifti_data_type * A, int m, int n){

	if (m < n){
		fprintf(stderr, "rows parameter (m) is smaller than columns parameter (n)\n");
		exit(EXIT_FAILURE);
	}

	culaStatus status;
	checkCudaErrors(hipSetDevice(0));

	//prepare arguments for cusolver svd
	char jobu = 'O';
	char jobvt = 'S';
	int lda = m; // leading dimension is equal to m ?? (or n ??)
    int ldu = m;
    int ldvt = n;

	nifti_data_type *S = (nifti_data_type*) malloc(imin(m,n) * sizeof(nifti_data_type));
    nifti_data_type *U = (nifti_data_type*) malloc(ldu*m* sizeof(nifti_data_type));
    nifti_data_type *VT = (nifti_data_type*) malloc(ldvt*n* sizeof(nifti_data_type));

	/* Initialize CULA */
    status = culaInitialize();
    checkStatus(status);

	/* Perform singular value decomposition CULA */
    printf("Performing singular value decomposition using CULA ... ");

	hipEvent_t start, stop;
	float elapsedTime;

	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	checkCudaErrors(hipEventRecord(start, 0));

    status = culaSgesvd(jobu, jobvt, m, n, A, lda, S, U, ldu, VT, ldvt);
    checkStatus(status);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));

	//copy results from gpu memory to cpu
	//checkCudaErrors(hipMemcpy(c, dev_A, m*n*sizeof(float), hipMemcpyDeviceToHost));
	//nifti_data_type * diagonalMatrix = (nifti_data_type *) malloc(imin(m,n)*sizeof(nifti_data_type));
	//checkCudaErrors(hipMemcpy(diagonalMatrix, S, imin(m,n)*sizeof(nifti_data_type), hipMemcpyDeviceToHost));
	//int k = imin(m,n);
	//free(diagonalMatrix);
	
	//free host memory
	free(S);
	free(U);
	free(VT);

	checkCudaErrors(hipDeviceReset()); // dla debuggera
	
	printf("Kernel-only time: %f ms\n", elapsedTime);

	return;
}