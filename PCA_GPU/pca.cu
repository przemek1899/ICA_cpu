#include "hip/hip_runtime.h"

/*
 * PCA Principal Component Analysis on raw data
 * This implementation bases on matlab pca implementation
 */

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include "hipsolver.h"
#include "pca.cuh"

#define imin(X, Y)  ((X) < (Y) ? (X) : (Y))

__global__ void pca_gpu(float* tab, int n){

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n){
		tab[i] = i*i;
	}

}

void checkCuSolverErrors(hipsolverStatus_t code){

	if(code){
		fprintf(stderr, "Cuda solver error code %d\n", static_cast<unsigned int>(code));
		hipDeviceReset();
        // Make sure we call CUDA Device Reset before exiting
        exit(EXIT_FAILURE);
	}
}

void runPCA(nifti_data_type * data, int m, int n){

	if (m < n){
		fprintf(stderr, "rows parameter (m) is smaller than columns parameter (n)\n");
		exit(EXIT_FAILURE);
	}

	checkCudaErrors(hipSetDevice(0));

	//initialize cusolverDn
	hipsolverHandle_t handle = NULL;
	hipsolverDnCreate(&handle); //sprawdzac checkCudaErrors

	//allocate memory
	nifti_data_type * dev_A;
    checkCudaErrors(hipMalloc(&dev_A, m*n*sizeof(nifti_data_type)));
    
	// copy data from cpu to gpu memory
    checkCudaErrors(hipMemcpy(dev_A, data, m*n*sizeof(nifti_data_type), hipMemcpyHostToDevice));

	// calculate the size needed for pre-allocated buffer
	// xy - numer of rows, zv - number of columns
	int Lwork;
	checkCuSolverErrors(hipsolverDnSgesvd_bufferSize(handle, m, n, &Lwork));

	//prepare arguments for cusolver svd
	char jobu = 'A';
	char jobvt = 'A';
	int *devInfo; checkCudaErrors(hipMalloc(&devInfo, sizeof(int)));
	int lda = m; // leading dimension is equal to m ?? (or n ??)
    int ldu = m;
    int ldvt = n;

	// below there are some notes from the cuda toolkit cusolver documentation
	// Note that the routine returns V H , not V.
	// Remark 1: gesvd only supports m>=n.  VEEEEEEEEERY IMPORTANT !!!!!!!!!!!!!!!!!!!!!
	// Remark 2: gesvd only supports jobu='A' and jobvt='A' and returns matrix U and V H .
	// rwork - needed for data types C,Z

	printf("m = %d, n = %d, Lwork = %d\n", m, n, Lwork);

	nifti_data_type * S, *U, *VT, *Work, *rwork;
	checkCudaErrors(hipMalloc(&S, imin(m,n)*sizeof(nifti_data_type)));
	checkCudaErrors(hipMalloc(&U, ldu*m*sizeof(nifti_data_type)));
	checkCudaErrors(hipMalloc(&VT, ldvt*n*sizeof(nifti_data_type)));
	checkCudaErrors(hipMalloc(&Work, Lwork*sizeof(nifti_data_type)));
	//checkCudaErrors(hipMalloc(&rwork, 5*imin(m,n)*sizeof(nifti_data_type)));

	// do we really need rwork??
	// run cusolver svd
	printf("before run cusolver svd\n");
	checkCuSolverErrors(hipsolverDnSgesvd(handle, jobu, jobvt, m, n, dev_A, lda, S, U, ldu, VT, ldvt, Work, Lwork, rwork, devInfo));
	int h_devInfo;
	hipDeviceSynchronize();
	checkCudaErrors(hipMemcpy(&h_devInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost));
	printf("devInfo %d\n", h_devInfo);

	hipEvent_t start, stop;
	float elapsedTime;

	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	checkCudaErrors(hipEventRecord(start, 0));

	// call kernel function here
	//pca_gpu<<<64, 64>>>(dev_A, m*n);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));

	//copy results from gpu memory to cpu
	//checkCudaErrors(hipMemcpy(c, dev_A, m*n*sizeof(float), hipMemcpyDeviceToHost));
	//nifti_data_type * diagonalMatrix = (nifti_data_type *) malloc(imin(m,n)*sizeof(nifti_data_type));
	//checkCudaErrors(hipMemcpy(diagonalMatrix, S, imin(m,n)*sizeof(nifti_data_type), hipMemcpyDeviceToHost));
	//int k = imin(m,n);
	//free(diagonalMatrix);
	
	//free gpu memory
	checkCudaErrors(hipFree(dev_A));
	checkCudaErrors(hipFree(S));
	checkCudaErrors(hipFree(U));
	checkCudaErrors(hipFree(VT));
	checkCudaErrors(hipFree(Work));
	checkCudaErrors(hipFree(rwork));
	checkCudaErrors(hipFree(devInfo));

	hipsolverDnDestroy(handle); //sprawdzac checkCudaErrors
	checkCudaErrors(hipDeviceReset()); // dla debuggera
	//free(c);
	
	printf("Kernel-only time: %f ms\n", elapsedTime);

	return;
}