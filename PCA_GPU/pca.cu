#include "hip/hip_runtime.h"

/*
 * PCA Principal Component Analysis on raw data
 * This implementation bases on matlab pca implementation
 */

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include "hipsolver.h"
#include "pca.cuh"

#define imin(X, Y)  ((X) < (Y) ? (X) : (Y))

__global__ void pca_gpu(float* tab, int n){

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n){
		tab[i] = i*i;
	}

}

void runPCA(nifti_data_type * data, int m, int n){

	checkCudaErrors(hipSetDevice(0));

	//initialize cusolverDn
	hipsolverHandle_t handle = NULL;
	hipsolverDnCreate(&handle); //sprawdzac checkCudaErrors

	//allocate memory
	nifti_data_type * dev_A;
    checkCudaErrors(hipMalloc(&dev_A, m*n*sizeof(nifti_data_type)));
    
	// copy data from cpu to gpu memory
    checkCudaErrors(hipMemcpy(dev_A, data, m*n*sizeof(nifti_data_type), hipMemcpyHostToDevice));

	// calculate the size needed for pre-allocated buffer
	// xy - numer of rows, zv - number of columns
	int Lwork;
	checkCudaErrors(hipsolverDnSgesvd_bufferSize(handle, m, n, &Lwork));

	//prepare arguments for cusolver svd
	char jobu = 'A';
	char jobvt = 'A';
	int *devInfo = NULL;
	int lda = m; // leading dimension is equal to m ?? (or n ??)
    int ldu = m;
    int ldvt = n;

	// below there are some notes from the cuda toolkit cusolver documentation
	// Note that the routine returns V H , not V.
	// Remark 1: gesvd only supports m>=n.  VEEEEEEEEERY IMPORTANT !!!!!!!!!!!!!!!!!!!!!
	// Remark 2: gesvd only supports jobu='A' and jobvt='A' and returns matrix U and V H .
	// rwork - needed for data types C,Z

	nifti_data_type * S, *U, *VT, *Work, *rwork;
	checkCudaErrors(hipMalloc(&S, imin(m,n)*sizeof(nifti_data_type)));
	checkCudaErrors(hipMalloc(&U, ldu*m*sizeof(nifti_data_type)));
	checkCudaErrors(hipMalloc(&VT, ldvt*n*sizeof(nifti_data_type)));
	checkCudaErrors(hipMalloc(&Work, Lwork*sizeof(nifti_data_type)));

	// do we really need rwork??
	// run cusolver svd
	printf("before run cusolver svd\n");
	checkCudaErrors(hipsolverDnSgesvd(handle, jobu, jobvt, m, n, dev_A, lda, S, U, ldu, VT, ldvt, Work, Lwork, rwork, devInfo));
	printf("after cusolver svd\n");

	hipEvent_t start, stop;
	float elapsedTime;

	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	checkCudaErrors(hipEventRecord(start, 0));

	// call kernel function here
	//pca_gpu<<<64, 64>>>(dev_A, m*n);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));

	//copy results from gpu memory to cpu
	//checkCudaErrors(hipMemcpy(c, dev_A, m*n*sizeof(float), hipMemcpyDeviceToHost));
	//nifti_data_type * diagonalMatrix = (nifti_data_type *) malloc(imin(m,n)*sizeof(nifti_data_type));
	//checkCudaErrors(hipMemcpy(diagonalMatrix, S, imin(m,n)*sizeof(nifti_data_type), hipMemcpyDeviceToHost));
	//int k = imin(m,n);
	//free(diagonalMatrix);
	
	//free gpu memory
	checkCudaErrors(hipFree(dev_A));
	checkCudaErrors(hipFree(S));
	checkCudaErrors(hipFree(U));
	checkCudaErrors(hipFree(VT));
	checkCudaErrors(hipFree(Work));

	hipsolverDnDestroy(handle); //sprawdzac checkCudaErrors
	checkCudaErrors(hipDeviceReset()); // dla debuggera
	//free(c);
	
	printf("Kernel-only time: %f ms\n", elapsedTime);

	return;
}