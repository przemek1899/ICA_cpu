#include "hip/hip_runtime.h"

/*
 * PCA Principal Component Analysis on raw data
 * This implementation bases on matlab pca implementation
 */

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""

__global__ void pca_gpu(float* tab, int n){

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n){
		tab[i] = i*i;
	}

}

void runPCA(void){

	checkCudaErrors(hipSetDevice(0));

	//initialize cusolverDn
	hipsolverHandle_t handle = NULL;
	checkCudaErrors(hipsolverDnCreate(&handle));

	int m = 64;
	int n = 64;
	float *dev_A;
	//allocate memory
    //checkCudaErrors(hipMalloc(&dev_A, m*n*sizeof(float)));
    //checkCudaErrors(hipMalloc(&dev_C, m*m*sizeof(float)));
	/*
	// copy data from cpu to gpu memory
    checkCudaErrors(hipMemcpy(dev_A, A, m*n*sizeof(float), hipMemcpyHostToDevice));
	*/

	hipEvent_t start, stop;
	float elapsedTime;

	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	checkCudaErrors(hipEventRecord(start, 0));

	// call kernel function here
	//pca_gpu<<<64, 64>>>(dev_A, m*n);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));

	
	//float * c;
	//c = (float *) malloc(m*n*sizeof(float));

	//copy results from gpu memory to cpu
	//checkCudaErrors(hipMemcpy(c, dev_A, m*n*sizeof(float), hipMemcpyDeviceToHost));
	
	//free gpu memory
	//checkCudaErrors(hipFree(dev_A));
	checkCudaErrors(hipDeviceReset()); // dla debuggera
	//free(c);
	
	checkCudaErrors(hipsolverDnDestroy(handle));
	printf("Kernel-only time: %f ms\n", elapsedTime);

	return;
}