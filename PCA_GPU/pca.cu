#include "hip/hip_runtime.h"

/*
 * PCA Principal Component Analysis on raw data
 * This implementation bases on matlab pca implementation
 */

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include "hipsolver.h"
#include "pca.cuh"

__global__ void pca_gpu(float* tab, int n){

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n){
		tab[i] = i*i;
	}

}

void runPCA(nifti_data_type * data, int xyzv){

	checkCudaErrors(hipSetDevice(0));

	//initialize cusolverDn
	hipsolverHandle_t handle = NULL;
	hipsolverDnCreate(&handle); //sprawdzac checkCudaErrors

	//allocate memory
	nifti_data_type * dev_A;
    checkCudaErrors(hipMalloc(&dev_A, xyzv*sizeof(nifti_data_type)));
    
	// copy data from cpu to gpu memory
    checkCudaErrors(hipMemcpy(dev_A, data, xyzv*sizeof(nifti_data_type), hipMemcpyHostToDevice));
	

	hipEvent_t start, stop;
	float elapsedTime;

	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	checkCudaErrors(hipEventRecord(start, 0));

	// call kernel function here
	//pca_gpu<<<64, 64>>>(dev_A, m*n);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));

	//copy results from gpu memory to cpu
	//checkCudaErrors(hipMemcpy(c, dev_A, m*n*sizeof(float), hipMemcpyDeviceToHost));
	
	//free gpu memory
	checkCudaErrors(hipFree(dev_A));
	hipsolverDnDestroy(handle); //sprawdzac checkCudaErrors
	checkCudaErrors(hipDeviceReset()); // dla debuggera
	//free(c);
	
	printf("Kernel-only time: %f ms\n", elapsedTime);

	return;
}